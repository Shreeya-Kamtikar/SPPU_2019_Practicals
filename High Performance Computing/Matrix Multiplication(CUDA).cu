
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 4 // Matrix dimension size

// CUDA kernel for matrix-vector multiplication
__global__

void matrixVecMul(int* A, int* B, int* C, int size) 
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < size) 
    {
        int sum = 0;
        for (int k = 0; k < size; k++) 
        {
            sum += A[row * size + k] * B[k];
        }
        C[row] = sum;
    }
}

// Function to initialize a matrix or vector with random values
void initialize(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        vector[i] = rand() % 10; // Random values between 0 and 9
    }
}

// Function to print a vector
void print(int* vector, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        printf("%d ", vector[i]);
    }
    printf("\n");
}

// Function to print a matrix
void printMatrix(int* matrix, int size) 
{
    for (int i = 0; i < size; i++) 
    {
        for (int j = 0; j < size; j++) 
        {
            printf("%d ", matrix[i * size + j]);
        }
    printf("\n");
    }
}

int main() 
{
    int* A, * B, * C;
    size_t matrixBytes = N * N * sizeof(int);
    size_t vectorBytes = N * sizeof(int);
    
    // Allocate memory for matrix and vectors
    A = (int*)malloc(matrixBytes);
    B = (int*)malloc(vectorBytes);
    C = (int*)malloc(vectorBytes);
    int* X, * Y, * Z;
    hipMalloc(&X, matrixBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    // Initialize matrix A and vector B
    initialize(A, N * N); // A is an NxN matrix
    initialize(B, N); // B is a vector of size N

    // Print matrix A and vector B
    printf("Matrix A:\n");
    printMatrix(A, N);
    printf("Vector B:\n");
    print(B, N);

    // Copy data from host to device
    hipMemcpy(X, A, matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    // Define the number of threads and blocks
    int threadsPerBlock = 16;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the matrix-vector multiplication kernel
    matrixVecMul<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    
    // Copy the result back to the host
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    // Print the result of the multiplication
    printf("Matrix-Vector multiplication result (C = A * B):\n");
    print(C, N);

    // Free allocated memory
    free(A);
    free(B);
    free(C);
    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}

//commands to run:
// vim multiply.cu
// nvcc multiply.cu -o multiply
// ./multiply